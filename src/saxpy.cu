#include "hip/hip_runtime.h"
#include "saxpy.cuh"

// #include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
// #include <hip/hip_runtime.h>

namespace cuda
{
namespace
{

#define CUDA_ASSERT(err_n) cuda_assert(err_n, true, __FILE__, __LINE__);
inline void cuda_assert(hipError_t err_n, bool terminate, const char* filename, int lineno)
{
    if (err_n == hipSuccess)
    {
        return;
    }

    std::cerr << filename << ": " << lineno << std::endl
              << hipGetErrorName(err_n) << ": " << std::endl
              << hipGetErrorString(err_n) << std::endl;

    if (terminate)
    {
        std::exit(EXIT_FAILURE);
    }
}

// __global__ void saxpy(size_t n, const float a, const float* x, float* y)
// {
//     size_t i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < n)
//     {
//         y[i] = a * x[i] + y[i];
//     }
// }

} // namespace

void print_info()
{
    // Show CUDA version.
    int driver_version  = -1;
    int runtime_version = -1;

    CUDA_ASSERT(hipDriverGetVersion(&driver_version));
    CUDA_ASSERT(hipRuntimeGetVersion(&runtime_version));

    std::cout << "CUDA driver version: " << driver_version << std::endl;
    std::cout << "CUDA runtime version: " << runtime_version << std::endl;

    // Show all CUDA devices and their properties.
    int num_devices = 0;
    hipGetDeviceCount(&num_devices);

    std::cout << "Found " << num_devices << " CUDA device(s)." << std::endl;
    for (int i = 0; i < num_devices; ++i)
    {
        hipDeviceProp_t device;
        hipGetDeviceProperties(&device, i);

        std::cout << "Device number: " << i << std::endl;
        std::cout << "  Device name: " << device.name << std::endl;
        std::cout << "  Compute capability: " << device.major << "." << device.minor << std::endl;
        std::cout << "  Device clock rate: " << device.clockRate / 1.0e3 << " MHz" << std::endl;
        std::cout << "  Device memory: " << static_cast<float>(device.totalGlobalMem) / 1.0e9 << " GB" << std::endl;
        std::cout << "  Memory clock rate (effective): " << device.memoryClockRate / 1.0e3 << " MHz" << std::endl;
        std::cout << "  Memory bus width: " << device.memoryBusWidth << " bit" << std::endl;
        std::cout << "  Memory bandwidth: " << 2.0 * device.memoryClockRate * (device.memoryBusWidth / 8) / 1.0e6
                  << " GB/s" << std::endl;
    }
}

} // namespace cuda